
#include <hip/hip_runtime.h>
// Tests the phases generated for a CUDA offloading target for different
// combinations of:
// - Number of gpu architectures;
// - Host/device-only compilation;
// - User-requested final phase - binary or assembly.

// Test single gpu architecture with complete compilation.
//
// Test CUDA NVPTX phases.
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN:   --no-offload-new-driver --cuda-gpu-arch=sm_30 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=BIN %s
//
// BIN-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (host-[[T]])
// BIN-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (host-[[T]])
// BIN-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-[[T]])
// BIN-DAG: [[P3:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (device-[[T]], [[ARCH:sm_30]])
// BIN-DAG: [[P4:[0-9]+]]: preprocessor, {[[P3]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH]])
// BIN-DAG: [[P5:[0-9]+]]: compiler, {[[P4]]}, ir, (device-[[T]], [[ARCH]])
// BIN-DAG: [[P6:[0-9]+]]: backend, {[[P5]]}, assembler, (device-[[T]], [[ARCH]])
// BIN-DAG: [[P7:[0-9]+]]: assembler, {[[P6]]}, object, (device-[[T]], [[ARCH]])
// BIN-DAG: [[P8:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda]]:[[ARCH]])" {[[P7]]}, object
// BIN-DAG: [[P9:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH]])" {[[P6]]}, assembler
// BIN-DAG: [[P10:[0-9]+]]: linker, {[[P8]], [[P9]]}, cuda-fatbin, (device-[[T]])
// BIN-DAG: [[P11:[0-9]+]]: offload, "host-[[T]] (powerpc64le-ibm-linux-gnu)" {[[P2]]}, "device-[[T]] ([[TRIPLE]])" {[[P10]]}, ir
// BIN-DAG: [[P12:[0-9]+]]: backend, {[[P11]]}, assembler, (host-[[T]])
// BIN-DAG: [[P13:[0-9]+]]: assembler, {[[P12]]}, object, (host-[[T]])
// BIN-DAG: [[P14:[0-9]+]]: linker, {[[P13]]}, image, (host-[[T]])

//
// Test single gpu architecture up to the assemble phase.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN:   --no-offload-new-driver --cuda-gpu-arch=sm_30 %s -S 2>&1 \
// RUN: | FileCheck -check-prefixes=ASM %s
// ASM-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (device-[[T]], [[ARCH:sm_30]])
// ASM-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH]])
// ASM-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-[[T]], [[ARCH]])
// ASM-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-[[T]], [[ARCH]])
// ASM-DAG: [[P4:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda]]:[[ARCH]])" {[[P3]]}, assembler
// ASM-DAG: [[P5:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (host-[[T]])
// ASM-DAG: [[P6:[0-9]+]]: preprocessor, {[[P5]]}, [[T]]-cpp-output, (host-[[T]])
// ASM-DAG: [[P7:[0-9]+]]: compiler, {[[P6]]}, ir, (host-[[T]])
// ASM-DAG: [[P8:[0-9]+]]: backend, {[[P7]]}, assembler, (host-[[T]])

//
// Test two gpu architectures with complete compilation.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN:   --no-offload-new-driver --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=BIN2 %s
// BIN2-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (host-[[T]])
// BIN2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (host-[[T]])
// BIN2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-[[T]])
// BIN2-DAG: [[P3:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (device-[[T]], [[ARCH1:sm_30]])
// BIN2-DAG: [[P4:[0-9]+]]: preprocessor, {[[P3]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH1]])
// BIN2-DAG: [[P5:[0-9]+]]: compiler, {[[P4]]}, ir, (device-[[T]], [[ARCH1]])
// BIN2-DAG: [[P6:[0-9]+]]: backend, {[[P5]]}, assembler, (device-[[T]], [[ARCH1]])
// BIN2-DAG: [[P7:[0-9]+]]: assembler, {[[P6]]}, object, (device-[[T]], [[ARCH1]])
// BIN2-DAG: [[P8:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda]]:[[ARCH1]])" {[[P7]]}, object
// BIN2-DAG: [[P9:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH1]])" {[[P6]]}, assembler
// BIN2-DAG: [[P10:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (device-[[T]], [[ARCH2:sm_35]])
// BIN2-DAG: [[P11:[0-9]+]]: preprocessor, {[[P10]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH2]])
// BIN2-DAG: [[P12:[0-9]+]]: compiler, {[[P11]]}, ir, (device-[[T]], [[ARCH2]])
// BIN2-DAG: [[P13:[0-9]+]]: backend, {[[P12]]}, assembler, (device-[[T]], [[ARCH2]])
// BIN2-DAG: [[P14:[0-9]+]]: assembler, {[[P13]]}, object, (device-[[T]], [[ARCH2]])
// BIN2-DAG: [[P15:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH2]])" {[[P14]]}, object
// BIN2-DAG: [[P16:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH2]])" {[[P13]]}, assembler
// BIN2-DAG: [[P17:[0-9]+]]: linker, {[[P8]], [[P9]], [[P15]], [[P16]]}, cuda-fatbin, (device-[[T]])
// BIN2-DAG: [[P18:[0-9]+]]: offload, "host-[[T]] (powerpc64le-ibm-linux-gnu)" {[[P2]]}, "device-[[T]] ([[TRIPLE]])" {[[P17]]}, ir
// BIN2-DAG: [[P19:[0-9]+]]: backend, {[[P18]]}, assembler, (host-[[T]])
// BIN2-DAG: [[P20:[0-9]+]]: assembler, {[[P19]]}, object, (host-[[T]])
// BIN2-DAG: [[P21:[0-9]+]]: linker, {[[P20]]}, image, (host-[[T]])

//
// Test two gpu architecturess up to the assemble phase.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN:   --no-offload-new-driver --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s -S 2>&1 \
// RUN: | FileCheck -check-prefixes=ASM2 %s
// ASM2-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (device-[[T]], [[ARCH1:sm_30]])
// ASM2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH1]])
// ASM2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-[[T]], [[ARCH1]])
// ASM2-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-[[T]], [[ARCH1]])
// ASM2-DAG: [[P4:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda]]:[[ARCH1]])" {[[P3]]}, assembler
// ASM2-DAG: [[P5:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (device-[[T]], [[ARCH2:sm_35]])
// ASM2-DAG: [[P6:[0-9]+]]: preprocessor, {[[P5]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH2]])
// ASM2-DAG: [[P7:[0-9]+]]: compiler, {[[P6]]}, ir, (device-[[T]], [[ARCH2]])
// ASM2-DAG: [[P8:[0-9]+]]: backend, {[[P7]]}, assembler, (device-[[T]], [[ARCH2]])
// ASM2-DAG: [[P9:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH2]])" {[[P8]]}, assembler
// ASM2-DAG: [[P10:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (host-[[T]])
// ASM2-DAG: [[P11:[0-9]+]]: preprocessor, {[[P10]]}, [[T]]-cpp-output, (host-[[T]])
// ASM2-DAG: [[P12:[0-9]+]]: compiler, {[[P11]]}, ir, (host-[[T]])
// ASM2-DAG: [[P13:[0-9]+]]: backend, {[[P12]]}, assembler, (host-[[T]])

//
// Test single gpu architecture with complete compilation in host-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN:   --no-offload-new-driver --cuda-gpu-arch=sm_30 %s --cuda-host-only 2>&1 \
// RUN: | FileCheck -check-prefixes=HBIN %s
// HBIN-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (host-[[T]])
// HBIN-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (host-[[T]])
// HBIN-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-[[T]])
// HBIN-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (host-[[T]])
// HBIN-DAG: [[P4:[0-9]+]]: assembler, {[[P3]]}, object, (host-[[T]])
// HBIN-DAG: [[P5:[0-9]+]]: linker, {[[P4]]}, image, (host-[[T]])
// HBIN-NOT: device
//
// Test single gpu architecture up to the assemble phase in host-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN:   --no-offload-new-driver --cuda-gpu-arch=sm_30 %s --cuda-host-only -S 2>&1 \
// RUN: | FileCheck -check-prefixes=HASM %s
// HASM-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (host-[[T]])
// HASM-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (host-[[T]])
// HASM-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-[[T]])
// HASM-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (host-[[T]])
// HASM-NOT: device

//
// Test two gpu architectures with complete compilation in host-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN:   --no-offload-new-driver --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-host-only 2>&1 \
// RUN: | FileCheck -check-prefixes=HBIN2 %s
// HBIN2-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (host-[[T]])
// HBIN2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (host-[[T]])
// HBIN2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-[[T]])
// HBIN2-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (host-[[T]])
// HBIN2-DAG: [[P4:[0-9]+]]: assembler, {[[P3]]}, object, (host-[[T]])
// HBIN2-DAG: [[P5:[0-9]+]]: linker, {[[P4]]}, image, (host-[[T]])
// HBIN2-NOT: device

//
// Test two gpu architectures up to the assemble phase in host-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN:   --no-offload-new-driver --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-host-only -S \
// RUN: 2>&1 | FileCheck -check-prefixes=HASM2 %s
// HASM2-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (host-[[T]])
// HASM2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (host-[[T]])
// HASM2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-[[T]])
// HASM2-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (host-[[T]])
// HASM2-NOT: device

//
// Test single gpu architecture with complete compilation in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN:   --no-offload-new-driver --cuda-gpu-arch=sm_30 %s --cuda-device-only 2>&1 \
// RUN: | FileCheck -check-prefixes=DBIN %s
// DBIN-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (device-[[T]], [[ARCH:sm_30]])
// DBIN-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH]])
// DBIN-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-[[T]], [[ARCH]])
// DBIN-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-[[T]], [[ARCH]])
// DBIN-DAG: [[P4:[0-9]+]]: assembler, {[[P3]]}, object, (device-[[T]], [[ARCH]])
// DBIN-DAG: [[P5:[0-9]+]]: offload, "device-[[T]] (nvptx64-nvidia-cuda:[[ARCH]])" {[[P4]]}, object
// DBIN-NOT: host
//
// Test single gpu architecture up to the assemble phase in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN:   --no-offload-new-driver --cuda-gpu-arch=sm_30 %s --cuda-device-only -S 2>&1 \
// RUN: | FileCheck -check-prefixes=DASM %s
// DASM-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (device-[[T]], [[ARCH:sm_30]])
// DASM-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH]])
// DASM-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-[[T]], [[ARCH]])
// DASM-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-[[T]], [[ARCH]])
// DASM-DAG: [[P4:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda]]:[[ARCH]])" {[[P3]]}, assembler
// DASM-NOT: host

//
// Test two gpu architectures with complete compilation in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN:   --no-offload-new-driver --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-device-only 2>&1 \
// RUN: | FileCheck -check-prefixes=DBIN2 %s
// DBIN2-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (device-[[T]], [[ARCH:sm_30]])
// DBIN2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH]])
// DBIN2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-[[T]], [[ARCH]])
// DBIN2-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-[[T]], [[ARCH]])
// DBIN2-DAG: [[P4:[0-9]+]]: assembler, {[[P3]]}, object, (device-[[T]], [[ARCH]])
// DBIN2-DAG: [[P5:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda]]:[[ARCH]])" {[[P4]]}, object
// DBIN2-DAG: [[P6:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (device-[[T]], [[ARCH2:sm_35]])
// DBIN2-DAG: [[P7:[0-9]+]]: preprocessor, {[[P6]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH2]])
// DBIN2-DAG: [[P8:[0-9]+]]: compiler, {[[P7]]}, ir, (device-[[T]], [[ARCH2]])
// DBIN2-DAG: [[P9:[0-9]+]]: backend, {[[P8]]}, assembler, (device-[[T]], [[ARCH2]])
// DBIN2-DAG: [[P10:[0-9]+]]: assembler, {[[P9]]}, object, (device-[[T]], [[ARCH2]])
// DBIN2-DAG: [[P11:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH2]])" {[[P10]]}, object
// DBIN2-NOT: host
//
// Test two gpu architectures up to the assemble phase in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN:   --no-offload-new-driver --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-device-only -S \
// RUN: 2>&1 | FileCheck -check-prefixes=DASM2 %s
// DASM2-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (device-[[T]], [[ARCH:sm_30]])
// DASM2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH]])
// DASM2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-[[T]], [[ARCH]])
// DASM2-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-[[T]], [[ARCH]])
// DASM2-DAG: [[P4:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda]]:[[ARCH]])" {[[P3]]}, assembler
// DASM2-DAG: [[P5:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (device-[[T]], [[ARCH2:sm_35]])
// DASM2-DAG: [[P6:[0-9]+]]: preprocessor, {[[P5]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH2]])
// DASM2-DAG: [[P7:[0-9]+]]: compiler, {[[P6]]}, ir, (device-[[T]], [[ARCH2]])
// DASM2-DAG: [[P8:[0-9]+]]: backend, {[[P7]]}, assembler, (device-[[T]], [[ARCH2]])
// DASM2-DAG: [[P9:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH2]])" {[[P8]]}, assembler
// DASM2-NOT: host

//
// Test the phases generated when using the new offloading driver.
//
// RUN: %clang -### --target=powerpc64le-ibm-linux-gnu -ccc-print-phases --offload-new-driver -fgpu-rdc \
// RUN:   --offload-arch=sm_52 --offload-arch=sm_70 %s 2>&1 | FileCheck --check-prefix=NEW-DRIVER-RDC %s
//      NEW-DRIVER-RDC: 0: input, "[[INPUT:.+]]", cuda
// NEW-DRIVER-RDC-NEXT: 1: preprocessor, {0}, cuda-cpp-output
// NEW-DRIVER-RDC-NEXT: 2: compiler, {1}, ir
// NEW-DRIVER-RDC-NEXT: 3: input, "[[INPUT]]", cuda, (device-cuda, sm_52)
// NEW-DRIVER-RDC-NEXT: 4: preprocessor, {3}, cuda-cpp-output, (device-cuda, sm_52)
// NEW-DRIVER-RDC-NEXT: 5: compiler, {4}, ir, (device-cuda, sm_52)
// NEW-DRIVER-RDC-NEXT: 6: backend, {5}, assembler, (device-cuda, sm_52)
// NEW-DRIVER-RDC-NEXT: 7: assembler, {6}, object, (device-cuda, sm_52)
// NEW-DRIVER-RDC-NEXT: 8: offload, "device-cuda (nvptx64-nvidia-cuda:sm_52)" {7}, object
// NEW-DRIVER-RDC-NEXT: 9: input, "[[INPUT]]", cuda, (device-cuda, sm_70)
// NEW-DRIVER-RDC-NEXT: 10: preprocessor, {9}, cuda-cpp-output, (device-cuda, sm_70)
// NEW-DRIVER-RDC-NEXT: 11: compiler, {10}, ir, (device-cuda, sm_70)
// NEW-DRIVER-RDC-NEXT: 12: backend, {11}, assembler, (device-cuda, sm_70)
// NEW-DRIVER-RDC-NEXT: 13: assembler, {12}, object, (device-cuda, sm_70)
// NEW-DRIVER-RDC-NEXT: 14: offload, "device-cuda (nvptx64-nvidia-cuda:sm_70)" {13}, object
// NEW-DRIVER-RDC-NEXT: 15: clang-offload-packager, {8, 14}, image, (device-cuda)
// NEW-DRIVER-RDC-NEXT: 16: offload, "host-cuda (powerpc64le-ibm-linux-gnu)" {2}, "device-cuda (powerpc64le-ibm-linux-gnu)" {15}, ir
// NEW-DRIVER-RDC-NEXT: 17: backend, {16}, assembler, (host-cuda)
// NEW-DRIVER-RDC-NEXT: 18: assembler, {17}, object, (host-cuda)
// NEW-DRIVER-RDC-NEXT: 19: clang-linker-wrapper, {18}, image, (host-cuda)

// RUN: %clang -### -target powerpc64le-ibm-linux-gnu -ccc-print-phases --offload-new-driver \
// RUN:   --offload-arch=sm_52 --offload-arch=sm_70 %s 2>&1 | FileCheck --check-prefix=NEW-DRIVER %s
//      NEW-DRIVER: 0: input, "[[CUDA:.+]]", cuda, (host-cuda)
// NEW-DRIVER-NEXT: 1: preprocessor, {0}, cuda-cpp-output, (host-cuda)
// NEW-DRIVER-NEXT: 2: compiler, {1}, ir, (host-cuda)
// NEW-DRIVER-NEXT: 3: input, "[[CUDA]]", cuda, (device-cuda, sm_52)
// NEW-DRIVER-NEXT: 4: preprocessor, {3}, cuda-cpp-output, (device-cuda, sm_52)
// NEW-DRIVER-NEXT: 5: compiler, {4}, ir, (device-cuda, sm_52)
// NEW-DRIVER-NEXT: 6: backend, {5}, assembler, (device-cuda, sm_52)
// NEW-DRIVER-NEXT: 7: assembler, {6}, object, (device-cuda, sm_52)
// NEW-DRIVER-NEXT: 8: offload, "device-cuda (nvptx64-nvidia-cuda:sm_52)" {7}, "device-cuda (nvptx64-nvidia-cuda:sm_52)" {6}, object
// NEW-DRIVER-NEXT: 9: input, "[[CUDA]]", cuda, (device-cuda, sm_70)
// NEW-DRIVER-NEXT: 10: preprocessor, {9}, cuda-cpp-output, (device-cuda, sm_70)
// NEW-DRIVER-NEXT: 11: compiler, {10}, ir, (device-cuda, sm_70)
// NEW-DRIVER-NEXT: 12: backend, {11}, assembler, (device-cuda, sm_70)
// NEW-DRIVER-NEXT: 13: assembler, {12}, object, (device-cuda, sm_70)
// NEW-DRIVER-NEXT: 14: offload, "device-cuda (nvptx64-nvidia-cuda:sm_70)" {13}, "device-cuda (nvptx64-nvidia-cuda:sm_70)" {12}, object
// NEW-DRIVER-NEXT: 15: linker, {8, 14}, cuda-fatbin, (device-cuda)
// NEW-DRIVER-NEXT: 16: offload, "host-cuda (powerpc64le-ibm-linux-gnu)" {2}, "device-cuda (nvptx64-nvidia-cuda)" {15}, ir
// NEW-DRIVER-NEXT: 17: backend, {16}, assembler, (host-cuda)
// NEW-DRIVER-NEXT: 18: assembler, {17}, object, (host-cuda)
// NEW-DRIVER-NEXT: 19: clang-linker-wrapper, {18}, image, (host-cuda)

// RUN: %clang -### --target=powerpc64le-ibm-linux-gnu -ccc-print-phases --offload-new-driver \
// RUN:   --offload-arch=sm_52 --offload-arch=sm_70 %s %S/Inputs/empty.cpp 2>&1 | FileCheck --check-prefix=NON-CUDA-INPUT %s

//      NON-CUDA-INPUT: 0: input, "[[CUDA:.+]]", cuda, (host-cuda)
// NON-CUDA-INPUT-NEXT: 1: preprocessor, {0}, cuda-cpp-output, (host-cuda)
// NON-CUDA-INPUT-NEXT: 2: compiler, {1}, ir, (host-cuda)
// NON-CUDA-INPUT-NEXT: 3: input, "[[CUDA]]", cuda, (device-cuda, sm_52)
// NON-CUDA-INPUT-NEXT: 4: preprocessor, {3}, cuda-cpp-output, (device-cuda, sm_52)
// NON-CUDA-INPUT-NEXT: 5: compiler, {4}, ir, (device-cuda, sm_52)
// NON-CUDA-INPUT-NEXT: 6: backend, {5}, assembler, (device-cuda, sm_52)
// NON-CUDA-INPUT-NEXT: 7: assembler, {6}, object, (device-cuda, sm_52)
// NON-CUDA-INPUT-NEXT: 8: offload, "device-cuda (nvptx64-nvidia-cuda:sm_52)" {7}, "device-cuda (nvptx64-nvidia-cuda:sm_52)" {6}, object
// NON-CUDA-INPUT-NEXT: 9: input, "[[CUDA]]", cuda, (device-cuda, sm_70)
// NON-CUDA-INPUT-NEXT: 10: preprocessor, {9}, cuda-cpp-output, (device-cuda, sm_70)
// NON-CUDA-INPUT-NEXT: 11: compiler, {10}, ir, (device-cuda, sm_70)
// NON-CUDA-INPUT-NEXT: 12: backend, {11}, assembler, (device-cuda, sm_70)
// NON-CUDA-INPUT-NEXT: 13: assembler, {12}, object, (device-cuda, sm_70)
// NON-CUDA-INPUT-NEXT: 14: offload, "device-cuda (nvptx64-nvidia-cuda:sm_70)" {13}, "device-cuda (nvptx64-nvidia-cuda:sm_70)" {12}, object
// NON-CUDA-INPUT-NEXT: 15: linker, {8, 14}, cuda-fatbin, (device-cuda)
// NON-CUDA-INPUT-NEXT: 16: offload, "host-cuda (powerpc64le-ibm-linux-gnu)" {2}, "device-cuda (nvptx64-nvidia-cuda)" {15}, ir
// NON-CUDA-INPUT-NEXT: 17: backend, {16}, assembler, (host-cuda)
// NON-CUDA-INPUT-NEXT: 18: assembler, {17}, object, (host-cuda)
// NON-CUDA-INPUT-NEXT: 19: input, "[[CPP:.+]]", c++, (host-cuda)
// NON-CUDA-INPUT-NEXT: 20: preprocessor, {19}, c++-cpp-output, (host-cuda)
// NON-CUDA-INPUT-NEXT: 21: compiler, {20}, ir, (host-cuda)
// NON-CUDA-INPUT-NEXT: 22: backend, {21}, assembler, (host-cuda)
// NON-CUDA-INPUT-NEXT: 23: assembler, {22}, object, (host-cuda)
// NON-CUDA-INPUT-NEXT: 24: clang-linker-wrapper, {18, 23}, image, (host-cuda)

//
// Test the phases using the new driver in LTO-mode.
//
// RUN: %clang -### -target powerpc64le-ibm-linux-gnu --offload-new-driver -ccc-print-phases \
// RUN:        --offload-arch=sm_70 --offload-arch=sm_52 -foffload-lto -fgpu-rdc -c %s 2>&1 \
// RUN: | FileCheck -check-prefix=LTO %s
//      LTO: 0: input, "[[INPUT:.+]]", cuda, (host-cuda)
// LTO-NEXT: 1: preprocessor, {0}, cuda-cpp-output, (host-cuda)
// LTO-NEXT: 2: compiler, {1}, ir, (host-cuda)
// LTO-NEXT: 3: input, "[[INPUT]]", cuda, (device-cuda, sm_52)
// LTO-NEXT: 4: preprocessor, {3}, cuda-cpp-output, (device-cuda, sm_52)
// LTO-NEXT: 5: compiler, {4}, ir, (device-cuda, sm_52)
// LTO-NEXT: 6: backend, {5}, lto-bc, (device-cuda, sm_52)
// LTO-NEXT: 7: offload, "device-cuda (nvptx64-nvidia-cuda:sm_52)" {6}, lto-bc
// LTO-NEXT: 8: input, "[[INPUT]]", cuda, (device-cuda, sm_70)
// LTO-NEXT: 9: preprocessor, {8}, cuda-cpp-output, (device-cuda, sm_70)
// LTO-NEXT: 10: compiler, {9}, ir, (device-cuda, sm_70)
// LTO-NEXT: 11: backend, {10}, lto-bc, (device-cuda, sm_70)
// LTO-NEXT: 12: offload, "device-cuda (nvptx64-nvidia-cuda:sm_70)" {11}, lto-bc
// LTO-NEXT: 13: clang-offload-packager, {7, 12}, image, (device-cuda)
// LTO-NEXT: 14: offload, "host-cuda (powerpc64le-ibm-linux-gnu)" {2}, "device-cuda (powerpc64le-ibm-linux-gnu)" {13}, ir
// LTO-NEXT: 15: backend, {14}, assembler, (host-cuda)
// LTO-NEXT: 16: assembler, {15}, object, (host-cuda)

//
// Test that the new driver does not create actions for invalid architectures.
//
// RUN: not %clang -### --target=powerpc64le-ibm-linux-gnu --offload-new-driver \
// RUN:        -ccc-print-phases --offload-arch=sm_999 -fgpu-rdc -c %s 2>&1 \
// RUN: | FileCheck -check-prefix=INVALID-ARCH %s
//      INVALID-ARCH: error: unsupported CUDA gpu architecture: sm_999
//      INVALID-ARCH: 0: input, "[[INPUT:.+]]", cuda
// INVALID-ARCH-NEXT: 1: preprocessor, {0}, cuda-cpp-output
// INVALID-ARCH-NEXT: 2: compiler, {1}, ir
// INVALID-ARCH-NEXT: 3: backend, {2}, assembler
// INVALID-ARCH-NEXT: 4: assembler, {3}, object
